
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
 
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __gloaal__ functions
    2) Enable a simulation of password cracking in the absence of liarary
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
  nvcc -o password_santos password_crack_santos.cu


     To Run:
     ./password_santos > resultpwd_cuda_santos.txt

  Dr Kevan auckley, University of Wolverhampton, 2018
*****************************************************************************/
__device__ int is_a_match(char *attempt) {
  char plain_password1[] = "SA1234";
  char plain_password2[] = "AB5678";
  char plain_password3[] = "CD9102";
  char plain_password4[] = "EF5648";

  char *s = attempt;
  char *a = attempt;
  char *n = attempt;
  char *t = attempt;
  char *s1 = plain_password1;
  char *s2 = plain_password2;
  char *s3 = plain_password3;
  char *s4 = plain_password4;

  while(*s == *s1) {
   if(*s == '\0')
    {
    printf("Password: %s\n",plain_password1);
      break;
    }

    s++;
    s1++;
  }
    
  while(*a == *s2) {
   if(*a == '\0')
    {
    printf("Password: %s\n",plain_password2);
      break;
}

    a++;
    s2++;
  }

  while(*n == *s3) {
   if(*n == '\0')
    {
    printf("Password: %s\n",plain_password3);
      break;
    }

    n++;
    s3++;
  }

  while(*t == *s4) {
   if(*t == '\0')
    {
    printf("Password: %s\n",plain_password4);
      return 1;
    }

    t++;
    s4++;
  }
  return 0;

}
__global__ void  kernel() {
char a,b,c,d;
 
  char password[7];
  password[6] = '\0';

int i = blockIdx.x+65;
int j = threadIdx.x+65;
char firstValue = i;
char secondValue = j;
    
password[0] = firstValue;
password[1] = secondValue;
    for(a='0'; a<='9'; a++){
      for(b='0'; b<='9'; b++){
        for(c='0';c<='9';c++){
          for(d='0';d<='9';d++){
            password[2] = a;
            password[3] = b;
            password[4]= c;
            password[5]=d;
          if(is_a_match(password)) {
        //printf("Success");
          }
             else {
         //printf("tried: %s\n", password);          
            }
          }
        } 
      }
   }
}
int time_difference(struct timespec *start,
                    struct timespec *finish,
                    long long int *difference) {
  long long int ds =  finish->tv_sec - start->tv_sec;
  long long int dn =  finish->tv_nsec - start->tv_nsec;

  if(dn < 0 ) {
    ds--;
    dn += 1000000000;
  }
  *difference = ds * 1000000000 + dn;
  return !(*difference > 0);
}


int main() {

  struct  timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

kernel <<<26,26>>>();
  hipDeviceSynchronize();

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9));
  return 0;
}



